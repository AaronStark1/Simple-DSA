// Matrix Multiplication in CUDA C++
// Can be scaled up for thousands of operations

#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

#define N 1024
#define PRINT_SIZE 8
#define COUNT 1

using namespace std;

__global__ void multiplication_kernel(float* A, float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {

    srand(static_cast<unsigned int>(time(0)));

    size_t size = N * N * sizeof(float);

    float *h_A = new float[N*N];
    float *h_B = new float[N*N];
    float *h_C = new float[N*N];

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int iter = 0;
    while (iter < COUNT) {

        for (int i = 0; i < N*N; ++i) {
            h_A[i] = static_cast<float>(rand()) / RAND_MAX;
            h_B[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        multiplication_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        // Output truncated to a 8 x 8 matrix
        cout << "\nTruncated output matrix (first " << PRINT_SIZE << "x" << PRINT_SIZE << "):\n";
        for (int i = 0; i < PRINT_SIZE; ++i) {
            for (int j = 0; j < PRINT_SIZE; ++j) {
                cout << h_C[i * N + j] << "\t";
            }
            cout << "\n";
        }

        iter++;
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;

    return 0;
}
